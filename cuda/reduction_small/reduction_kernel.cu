#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
    Parallel reduction kernels
*/

#ifndef _REDUCE_KERNEL_H_
#define _REDUCE_KERNEL_H_

#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double>
{
    __device__ inline operator       double *()
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};

/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize>
__global__ void
reduce6(T *g_idata, T *g_odata, unsigned int n)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (i + blockSize < n)
            mySum += g_idata[i+blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    cg::sync(cta);


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    cg::sync(cta);

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

    cg::sync(cta);

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    cg::sync(cta);

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    if (cta.thread_rank() < 32)
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = tile32.size()/2; offset > 0; offset /= 2)
        {
             mySum += tile32.shfl_down(mySum, offset);
        }
    }

    // write result for this block to global mem
    if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void
reduce(int size, int threads, int blocks,
       T *d_idata, T *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

    switch (threads)
    {
        case 512:
            reduce6<T, 512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case 256:
            reduce6<T, 256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case 128:
            reduce6<T, 128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case 64:
            reduce6<T,  64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case 32:
            reduce6<T,  32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case 16:
            reduce6<T,  16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case  8:
            reduce6<T,   8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case  4:
            reduce6<T,   4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case  2:
            reduce6<T,   2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;

        case  1:
            reduce6<T,   1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
            break;
    }
}

// Instantiate the reduction function for 3 types
template void
reduce<int>(int size, int threads, int blocks,
            int *d_idata, int *d_odata);

template void
reduce<float>(int size, int threads, int blocks,
              float *d_idata, float *d_odata);

template void
reduce<double>(int size, int threads, int blocks,
               double *d_idata, double *d_odata);


#endif // #ifndef _REDUCE_KERNEL_H_
