#include "hip/hip_runtime.h"

#include "main.cuh"

#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;

int main(int argc, char **argv)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 1 << 19;
    cout << "Vector addition of " << numElements << " elements" << endl;

    // host memory
    float *h_A = NULL;
    float *h_B = NULL;
    float *h_C = NULL;

    // device memory
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;

    if (!AlocateHstMemory(&h_A, numElements, "h_A")) exit(1);
    if (!AlocateHstMemory(&h_B, numElements, "h_B")) exit(1);
    if (!AlocateHstMemory(&h_C, numElements, "h_C")) exit(1);

    RandInit(h_A, numElements);
    RandInit(h_B, numElements);

    if (!AlocateDevMemory(&d_A, numElements, "d_A")) exit(1);
    if (!AlocateDevMemory(&d_B, numElements, "d_B")) exit(1);
    if (!AlocateDevMemory(&d_C, numElements, "d_C")) exit(1);

    // Copy input to the device
    CopyHst2DevMemory(h_A, d_A, numElements);
    CopyHst2DevMemory(h_B, d_B, numElements);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

    cout << "CUDA kernel launch with " << blocksPerGrid;
    cout << " blocks of " << threadsPerBlock << " threads" << endl;

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        cout << "status: " << hipGetErrorString(err) << endl;
        cout << "Failed to launch vectorAdd kernel" << endl;
        exit(1);
    }

    // Copy results to the host
    CopyDev2HstMemory(d_C, h_C, numElements);

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            cout << "Result verification failed at element " << i << endl;
            exit(1);
        }
    }

    cout << "Test PASSED" << endl;

    // Free device global memory
    freeDev(d_A);
    freeDev(d_B);
    freeDev(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    cout << "Done" << endl;
    cout << endl;

    return 0;
}

// ---------------------------------------------------------------------------
// Alocate host memory and perform validation.
// ---------------------------------------------------------------------------
bool AlocateHstMemory(float** h, const int& numElements, const string& name)
{
    size_t size = numElements * sizeof(float);

    *h = (float *)malloc(size);

    if (*h != NULL)
        return true;

    cout << "Failed to allocate host memory: " << name << endl;

    return false;
}

// ---------------------------------------------------------------------------
// Alocate device memory and perform validation.
// ---------------------------------------------------------------------------
bool AlocateDevMemory(float** d, const int& numElements, const string& name)
{
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(float);

    err = hipMalloc((void **)d, size);

    if (err == hipSuccess)
        return true;

    cout << "status: " << hipGetErrorString(err) << endl;
    cout << "Failed to allocate device memory: " << name << endl;

    return false;
}

// ---------------------------------------------------------------------------
// Initialize host memory.
// ---------------------------------------------------------------------------
void RandInit(float* h, const int& numElements)
{
    for (int i = 0; i < numElements; ++i)
        h[i] = rand()/(float)RAND_MAX;
}

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
bool CopyHst2DevMemory(float* h, float* d, const int& numElements)
{
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(float);

    cout << "Copy host memory to the CUDA device." << endl;
    err = hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    if (err == hipSuccess)
        return true;

    cout << "status: " << hipGetErrorString(err) << endl;
    cout << "Failed to copy host memory to the CUDA device." << endl;

    return false;
}

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
bool CopyDev2HstMemory(float* d, float* h, const int& numElements)
{
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(float);

    cout << "Copy device memory to the host." << endl;
    err = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

    if (err == hipSuccess)
        return true;

    cout << "status: " << hipGetErrorString(err) << endl;
    cout << "Failed to copy device memory to the host." << endl;

    return false;
}

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
bool freeDev(float* d_A)
{
    hipError_t err = hipSuccess;

    err = hipFree(d_A);

    if (err == hipSuccess)
        return true;

    cout << "status: " << hipGetErrorString(err) << endl;
    cout << "Failed to free device vector A" << endl;

    return false;
}
