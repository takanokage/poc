#include "hip/hip_runtime.h"

#include "vectorAdd.cuh"

// ---------------------------------------------------------------------------
// C = A + B
// ---------------------------------------------------------------------------
__global__ void vectorAdd(
    const float* A,
    const float* B,
    float* const C,
    int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
        C[i] = A[i] + B[i];
}
